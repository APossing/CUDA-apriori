#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>
#include <stdio.h>
#include "CUDABackground.h"
#include "FileReader.h"
#include <atomic>
#include <iostream>
#include <pplinterface.h>
using namespace std;

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);
hipError_t findFrequents(int**main, unsigned int *counts, int mainSize, int countsSize);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

__global__ void findFrequents(int**mainArray, unsigned int *counts)
{
	int i = threadIdx.x;
	//int end = mainArray[i][0] + 1;
	printf("%d:\t%d\n", threadIdx.x, mainArray[i][1]);
	int * startAddy = mainArray[i];
	for(int j = 1; j <= 1; j++)
	{
		//atomicAdd(counts + mainArray[i][j], 1);
	}
}

int main()
{

	CUDABackground cuda = CUDABackground();
	FileReader f = FileReader("data.txt");
	int cudaCores = cuda.calculateCores();
	unsigned int * frequency = (unsigned int *)malloc(sizeof(unsigned int) * f.maxNumber);
	for (int i = 0; i < f.maxNumber; i++)
	{
		frequency[i] = 0;
	}
	for (int i = 0; i < 50; i++)
	{
		printf("\n%d: ", i);
		for (int j = 0; j < f.master[i][0];j++)
			printf("%d,", f.master[i][j]);
	}
	cout<<frequency[0];
	1 + 1;
	hipError_t cudaStatus = findFrequents(f.master, frequency, f.count, f.maxNumber);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}
	int test2 = frequency[0];




    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
    //hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

hipError_t findFrequents(int* main[], unsigned int *counts, int mainSize, int countsSize)
{
	int **devMain = 0;
	unsigned int * devCounts = 0;
	hipError_t cudaStatus;
	cudaStatus = hipMalloc((void***)&devMain, mainSize * sizeof(int*));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	for (int i = 0; i < mainSize; i++)
	{
		int* temp;
		hipMalloc((void**)&temp, sizeof(int)* (main[i][0] + 1)); // allocate for 1 int in each int pointer
		hipMemcpy(temp, main[i], sizeof(int) * (main[i][0] + 1), hipMemcpyHostToDevice); // copy data
		hipMemcpy(devMain + i, &temp, sizeof(int*), hipMemcpyHostToDevice);
	}


	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&devMain, mainSize * sizeof(int*));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}


	cudaStatus = hipMalloc((void**)&devCounts, countsSize * sizeof(unsigned int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(devCounts, counts, countsSize * sizeof(unsigned int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	findFrequents << <1, 500 >> > (devMain, devCounts);


	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(counts, devCounts, countsSize * sizeof(unsigned int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(devMain);
	hipFree(counts);

	return cudaStatus;

}


// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
